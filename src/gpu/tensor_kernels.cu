// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// The MIT License (MIT)
//
// Copyright (c) 2019 Roberto Paredes Palacios, <rparedes@dsic.upv.es>

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

///////////////////////////////////////////
__global__ void sum_mat_row(float* a, float* b, float* c, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   c[thread_id_x]=a[thread_id_x]+b[thread_id_x%cols];

}
///////////////////////////////////////////
__global__ void sum_mat_col(float* a, float* b, float* c, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   c[thread_id_x]=a[thread_id_x]+b[thread_id_x/cols];

}
///////////////////////////////////////////
__global__ void set(float* a, float v, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=v;

}

///////////////////////////////////////////
__global__ void mult(float* a, float v, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]*=v;

}
///////////////////////////////////////////
__global__ void el_mult(float* a, float *b, float *c, int incC, int rows, int cols)
{
  int ops=rows*cols;
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (incC) c[thread_id_x]+=a[thread_id_x]*b[thread_id_x];
    else c[thread_id_x]=a[thread_id_x]*b[thread_id_x];
}

///////////////////////////////////////////
__global__ void el_div(float* a, float *b, float *c, int incC, int rows, int cols)
{
  int ops=rows*cols;
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (incC) c[thread_id_x]+=a[thread_id_x]/b[thread_id_x];
    else c[thread_id_x]=a[thread_id_x]/b[thread_id_x];
}

///////////////////////////////////////////
__global__ void sum(float* a, float v, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]+=v;

}
///////////////////////////////////////////
__global__ void sum(float scA,float* a,float scB,float *b, float *c,int incC, int tam)
{
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < tam) {
    if (incC) c[thread_id_x]+=scA*a[thread_id_x]+scB*b[thread_id_x];
    else c[thread_id_x]=scA*a[thread_id_x]+scB*b[thread_id_x];
  }
}

///////////////////////////////////////////
__global__ void reduce_array_sum(float* array, long int ops, int cols,float* result)
{
  extern __shared__ float arr_acc[];
  __shared__ float accumulate_result[1];

  int thread_id_x = threadIdx.x +blockIdx.x*blockDim.x;
  float sum=0;
  arr_acc[thread_id_x]=0.0;

  if(thread_id_x==0)
  	accumulate_result[thread_id_x]=0.0;

  __syncthreads();
  if (thread_id_x<ops)
  {
  	for (int i=0; i<cols;i++)
    		sum+=array[thread_id_x*cols+i];

  __syncthreads();
    	arr_acc[thread_id_x]=sum;
  __syncthreads();

  }

  if (thread_id_x==0)
  {
  	for (int i=0; i<ops;i++)
      accumulate_result[thread_id_x]+=arr_acc[thread_id_x+i];

    result[thread_id_x]=accumulate_result[thread_id_x];//copy back to global memory from shared

  }
}

///////////////////////////////////////////
__global__ void log(float* a, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=log(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void exp(float* a, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=exp(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void sqrt(float* a, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=sqrt(a[thread_id_x]);

}

///////////////////////////////////////////
__global__ void sqr(float* a, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]*=a[thread_id_x];

}

///////////////////////////////////////////
__global__ void mask(float* a, float v, int rows, int cols)
{
 int ops=rows*cols;
 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < ops)
   a[thread_id_x]=a[thread_id_x]<v;

}

///////////////////////////////////////////

__global__ void reduce_sum2D(float *a,float *b,int rows,int cols,int axis)
{
  int ops=rows*cols;
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < ops)
    if (axis==0)
        b[thread_id_x%cols]+=a[thread_id_x];
    else
        b[thread_id_x/cols]+=a[thread_id_x];
}
///////////////////////////////////////////
__global__ void cent(float* a, float* b, float* c, int tam)
{

 int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < tam){
   c[thread_id_x]=0;
   if (a[thread_id_x]) c[thread_id_x]+=a[thread_id_x]*log(b[thread_id_x]);
   if (a[thread_id_x]!=1.0) c[thread_id_x]+=(1.0-a[thread_id_x])*log(1.0-b[thread_id_x]);
  }
}


__global__ void accuracy(float* T, float* N,float* acc,int cols, long int total_ops, int* MC_err)
{

int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
int result_t=T[thread_id_x*cols];
float result_n=N[thread_id_x*cols];

int row_max_t=0;
int row_max_n=0;

int aux_t;
float aux_n;
if (thread_id_x < total_ops)
{
  for(int i=1;i<cols;i++)
  {
   aux_t=T[thread_id_x*cols+i];
   aux_n=N[thread_id_x*cols+i];

	if (aux_t>result_t)
	 {
  		result_t=aux_t;
      row_max_t=i;
   }
  if (aux_n>result_n)
	 {
		result_n=aux_n;
    row_max_n=i;
   }
  }

  acc[thread_id_x]=row_max_t;
  atomicAdd(MC_err,(int)(row_max_t==row_max_n));
}

}

///////////////////////////////////////////
__global__ void relu(float *a,float *b,int tam)
{
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < tam){
    if (a[thread_id_x]>0.0) b[thread_id_x]=a[thread_id_x];
    else b[thread_id_x]=0.0;
   }
}


__global__ void d_relu(float *d,float *i,float *pd,int tam)
{
  int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x < tam){
    if (i[thread_id_x]>0.0) pd[thread_id_x]=d[thread_id_x];
    else pd[thread_id_x]=0.0;
   }

}

///////////////////////////////////////////
__global__ void softmax(float* E,float* N,float* auxE ,long int sample_dim, long int n_vals)
{
    float C_value=0;
    int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
    float maxCoef = E[thread_id_x*sample_dim];
    float actualCoef = 0;
    if (thread_id_x<n_vals)
    {

	    for (int cA = 1; cA < sample_dim; cA++)
    		if (E[thread_id_x*sample_dim+cA] > maxCoef)
    			 maxCoef=E[thread_id_x*sample_dim+cA];

	    for (int cA = 0; cA < sample_dim; cA++)
  		{
  			actualCoef=expf(E[thread_id_x*sample_dim+cA]-maxCoef);
  			auxE[thread_id_x*sample_dim+cA]=actualCoef;
        C_value+=actualCoef;
  		}

      for (int cA=0; cA < sample_dim; cA++)
	       N[thread_id_x*sample_dim+cA]=auxE[thread_id_x*sample_dim+cA]/C_value;
    }

}
























///////////////////////////////////////////


///////////////////////////////////////////


///////////////////////////////////////////
