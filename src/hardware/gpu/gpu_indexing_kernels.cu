/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void gpu_where(float *condition, float *A, float *B, float *C, long int size){
    long int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id_x < size){
        if((bool) condition[thread_id_x]){
            C[thread_id_x] = A[thread_id_x];
        }else{
            C[thread_id_x] = B[thread_id_x];
        }
    }
}
