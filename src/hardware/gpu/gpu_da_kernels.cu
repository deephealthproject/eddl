#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#define _USE_MATH_DEFINES
#include <cmath>
#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "eddl/hardware/gpu/gpu_kernels.h"

__device__ void gpu_single_shift(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, int* shift, int mode, float constant){
    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int *B_stride = A_stride;

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % irows;
    int Bj = thread_id_x / B_stride[3] % icols;
    //--------------
    //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

    int Ai = Bi - shift[0];
    int Aj = Bj - shift[1];

    if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }else{
        if(mode==0){ // Constant
            B[thread_id_x] = constant;
        }else if(mode == 5){  // Original
            B[thread_id_x] = A[thread_id_x];
        }else{
            printf("Mode (%d) not implemented (%s)", mode, "Tensor::gpu_single_shift");
        }
    }
}


__device__ void gpu_single_rotate(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, float angle_rad, int* center, int mode, float constant){
    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int *B_stride = A_stride;

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % irows;
    int Bj = thread_id_x / B_stride[3] % icols;
    //--------------
    //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

    int Bi_c = Bi - center[0];
    int Bj_c = Bj - center[1];
    int Ai = sinf(angle_rad) * Bj_c + cosf(angle_rad) * Bi_c + center[0];
    int Aj = cosf(angle_rad) * Bj_c - sinf(angle_rad) * Bi_c + center[1];

    if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }else{
        if(mode==0){ // Constant
            B[thread_id_x] = constant;
        }else if(mode == 5){  // Original
            B[thread_id_x] = A[thread_id_x];
        }else{
            printf("Mode (%d) not implemented (%s)\n", mode, "Tensor::gpu_single_rotate");
        }
    }
}


__device__ void gpu_single_scale(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* new_shape, int mode, float constant){
    int offsets[2] = {0, 0};
    offsets[0] = (new_shape[0] - orows)/2.0f;
    offsets[1] = (new_shape[1] - ocols)/2.0f;

    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % orows;
    int Bj = thread_id_x / B_stride[3] % ocols;
    //--------------
    //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);


    int Ai = ((Bi + offsets[0]) * irows) / new_shape[0];
    int Aj = ((Bj + offsets[1]) * icols) / new_shape[1];

    if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols) {
        int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
        B[thread_id_x] = A[A_pos];
    } else {
        if(mode==0){ // Constant
            B[thread_id_x] = constant;
        }else if(mode == 5){  // Original
            B[thread_id_x] = A[thread_id_x];
        }else{
            printf("Mode (%d) not implemented (%s)", mode, "Tensor::gpu_single_scale");
        }
    }
}



__device__ void gpu_single_flip(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, int axis, bool apply){
    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int *B_stride = A_stride;

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % irows;
    int Bj = thread_id_x / B_stride[3] % icols;
    //--------------
    //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

    if(apply){
        int pos[2] = {Bi, Bj}; pos[axis] = (irows-1) - pos[axis];
        int Ai = pos[0]; int Aj = pos[1];
        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }else{
        B[thread_id_x] = A[thread_id_x];
    }
}


__device__ void gpu_single_crop(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int* offsets, float constant, bool inverse){
    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % orows;
    int Bj = thread_id_x / B_stride[3] % ocols;

    // Compute coordinates
    int Ai = Bi + offsets[0];  // Start from the (0,0) of the cropping area
    int Aj = Bj + offsets[1];

    bool inRegion = Ai >= coords_from[0] && Ai <= coords_to[0] && Aj >= coords_from[1] && Aj <= coords_to[1];
    if ((inRegion && !inverse) || (!inRegion && inverse)){
        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }else{
        B[thread_id_x] = constant;
    }
}


__device__ void gpu_single_crop_scale(long int thread_id_x, float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int mode, float constant){
    int A_hc = coords_to[0]-coords_from[0]+1;
    int A_wc = coords_to[1]-coords_from[1]+1;

    int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
    int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

    //--------------
    int b = thread_id_x / B_stride[0] % batch;
    int c = thread_id_x / B_stride[1] % depth;
    int Bi = thread_id_x / B_stride[2] % orows;
    int Bj = thread_id_x / B_stride[3] % ocols;

    int Ai = (Bi * A_hc) / orows + coords_from[0];
    int Aj = (Bj * A_wc) / ocols + coords_from[1];

    int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
    B[thread_id_x] = A[A_pos];
}


__global__ void shift(float* A, float* B, int batch, int depth, int irows, int icols, int* shift, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        gpu_single_shift(thread_id_x, A, B, batch, depth, irows, icols, shift, mode, constant);
    }

}


__global__ void rotate(float* A, float* B, int batch, int depth, int irows, int icols, float angle_rad, int* center, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    // Not implemented
    if (thread_id_x < ops){
        gpu_single_rotate(thread_id_x, A, B, batch, depth, irows, icols, angle_rad, center, mode, constant);
    }
}


__global__ void scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* new_shape, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        gpu_single_scale(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, new_shape, mode, constant);
    }

}


__global__ void flip(float* A, float* B, int batch, int depth, int irows, int icols, int axis){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        gpu_single_flip(thread_id_x, A, B, batch, depth, irows, icols, axis, true);
    }
}


__global__ void crop(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int* offsets, float constant, bool inverse){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        gpu_single_crop(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, coords_from, coords_to, offsets, constant, inverse);
    }
}


__global__ void crop_scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        gpu_single_crop_scale(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, coords_from, coords_to, mode, constant);
    }
}


__global__ void shift_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor_x, float* factor_y, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int b = thread_id_x / (depth*irows*icols) % batch;

        int shift_y = (int)(irows * ((factor_y[1]-factor_y[0]) * rnd[b+1] + factor_y[0]));
        int shift_x = (int)(icols * ((factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]));
        int shift[2] = {shift_y, shift_x};

        gpu_single_shift(thread_id_x, A, B, batch, depth, irows, icols, shift, mode, constant);
    }

}

__global__ void rotate_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor, int* center, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int b = thread_id_x / (depth*irows*icols) % batch;

        float angle = -1.0f * ((factor[1]-factor[0]) * rnd[b] + factor[0]);
        float angle_rad = (float)((-angle) * M_PI/180.0f);  // Convert to radians

        gpu_single_rotate(thread_id_x, A, B, batch, depth, irows, icols, angle_rad, center, mode, constant);
    }
}

__global__ void scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        int b = thread_id_x / (depth*orows*ocols) % batch;

        float scale = (factor[1]-factor[0]) * rnd[b] + factor[0];
        int new_shape_y = (int)(irows * scale);
        int new_shape_x = (int)(icols * scale);
        int new_shape[2] = {new_shape_y, new_shape_x};

        gpu_single_scale(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, new_shape, mode, constant);
    }

}


__global__ void flip_random(float* A, float* B, int batch, int depth, int irows, int icols, int axis, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        //--------------
        int b = thread_id_x / (depth*irows*icols) % batch;

        bool apply = rnd[b] >= 0.5f;
        gpu_single_flip(thread_id_x, A, B, batch, depth, irows, icols, axis, apply);
    }
}


__global__ void crop_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int b = thread_id_x / (depth*orows*ocols) % batch;

        // Compute random coordinates
        int w = ocols;
        int h = orows;
        int x = (int)((icols-w) * rnd[b]);
        int y = (int)((irows-h) * rnd[b+1]);

        int coords_from_x = x;
        int coords_to_x = x+w;
        int coords_from_y = y;
        int coords_to_y = y+h;

        int coords_from[2] = {coords_from_y, coords_from_x};
        int coords_to[2] = {coords_to_y, coords_to_x};

        // Compute offsets
        int offsets[2] = {0, 0}; // Used only during the normal crop
        if(irows!=orows || icols!=ocols){
            offsets[0] = coords_from[0];
            offsets[1] = coords_from[1];
        }

        gpu_single_crop(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, coords_from, coords_to, offsets, 0.0f, false);
    }
}


__global__ void crop_scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor, int mode, float constant, float* rnd) {
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        //--------------
        int b = thread_id_x / (depth*orows*ocols) % batch;

        // Compute random coordinates
        float scale = ((factor[1]-factor[0]) * rnd[b] + factor[0]);
        int h = (int)(irows * scale);
        int w = (int)(icols * scale);
        int y = (int)((irows-h) * rnd[b+1]);
        int x = (int)((icols-w) * rnd[b+2]);

        int coords_from_x = x;
        int coords_to_x = x+w;
        int coords_from_y = y;
        int coords_to_y = y+h;

        int coords_from[2] = {coords_from_y, coords_from_x};
        int coords_to[2] = {coords_to_y, coords_to_x};

        gpu_single_crop_scale(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, coords_from, coords_to, mode, constant);
    }
}


__global__ void cutout_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor_x, float* factor_y, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int b = thread_id_x / (depth*orows*ocols) % batch;

        // Compute random coordinates
        int h = (int)(irows * ((factor_y[1]-factor_y[0]) * rnd[b] + factor_y[0]));
        int w = (int)(icols * ((factor_x[1]-factor_x[0]) * rnd[b+1] + factor_x[0]));
        int y = (int)((irows-h) * rnd[b+2]);
        int x = (int)((icols-w) * rnd[b+3]);

        int coords_from_x = x;
        int coords_to_x = x+w;
        int coords_from_y = y;
        int coords_to_y = y+h;

        int offsets[2] = {0, 0}; // Used only during the normal crop
        int coords_from[2] = {coords_from_y, coords_from_x};
        int coords_to[2] = {coords_to_y, coords_to_x};

        gpu_single_crop(thread_id_x, A, B, batch, depth, irows, icols, orows, ocols, coords_from, coords_to, offsets, constant, true);
    }
}
