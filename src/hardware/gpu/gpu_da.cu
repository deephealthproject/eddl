/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"

#ifndef M_PI
    #define M_PI 3.14159265358979323846
#endif


void gpu_shift(Tensor *A, Tensor *B, vector<int> t_shift, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(B);

    // Copy vector from host to device
    int *d_shift; hipMalloc((int**)&d_shift, 2*sizeof(int));
    hipMemcpy(d_shift, t_shift.data(), 2*sizeof(int), hipMemcpyHostToDevice);

    shift<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], d_shift, wrapping_mode, constant);
    check_cuda(hipDeviceSynchronize(), "shift");
}

void gpu_rotate(Tensor *A, Tensor *B, float angle, vector<int> offset_center, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Compute angle (radians) and center
    float side_a = A->shape[2]/2.0f;
    float side_b = A->shape[3]/2.0f;
    int center[2] = {(int)side_a+offset_center[0], (int)side_b+offset_center[1]};
    float angle_rad = (float)((-angle) * M_PI/180.0f);  // Convert to radians

    // Copy vector from host to device
    int *d_center; hipMalloc((int**)&d_center, 2*sizeof(int));
    hipMemcpy(d_center, center, 2*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    rotate<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], angle_rad, d_center, wrapping_mode, constant);
    check_cuda(hipDeviceSynchronize(), "rotate");

}

void gpu_scale(Tensor *A, Tensor *B, vector<int> new_shape, int wrapping_mode, float constant, int transformation_mode){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    int *d_new_shape; hipMalloc((int**)&d_new_shape, new_shape.size()*sizeof(int));
    hipMemcpy(d_new_shape, new_shape.data(), new_shape.size()*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    scale<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_new_shape, wrapping_mode, constant, transformation_mode);
    check_cuda(hipDeviceSynchronize(), "scale");
}

void gpu_scale_back(Tensor *A, Tensor *B, vector<int> new_shape, int wrapping_mode, float constant, int transformation_mode){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    int *d_new_shape; hipMalloc((int**)&d_new_shape, new_shape.size()*sizeof(int));
    hipMemcpy(d_new_shape, new_shape.data(), new_shape.size()*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    scale_back<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_new_shape, wrapping_mode, constant, transformation_mode);
    check_cuda(hipDeviceSynchronize(), "scale_back");
}


void gpu_flip(Tensor *A, Tensor *B, int axis){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(B);

    flip<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], axis);
    check_cuda(hipDeviceSynchronize(), "flip");
}

void gpu_crop(Tensor *A, Tensor *B, vector<int> coords_from, vector<int> coords_to, float constant, bool inverse){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(B);

    // Copy vector from host to device
    int *d_coords_from; hipMalloc((int**)&d_coords_from, coords_from.size()*sizeof(int));
    hipMemcpy(d_coords_from, coords_from.data(), coords_from.size()*sizeof(int), hipMemcpyHostToDevice);

    // Copy vector from host to device
    int *d_coords_to; hipMalloc((int**)&d_coords_to, coords_to.size()*sizeof(int));
    hipMemcpy(d_coords_to, coords_to.data(), coords_to.size()*sizeof(int), hipMemcpyHostToDevice);

    // Compute offsets
    int offsets[2] = {0, 0};
    if(!Tensor::sameShape(A, B)){
        offsets[0] = coords_from[0];
        offsets[1] = coords_from[1];
    }
    int *d_offsets; hipMalloc((int**)&d_offsets, 2*sizeof(int));
    hipMemcpy(d_offsets, offsets, 2*sizeof(int), hipMemcpyHostToDevice);

    crop<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_coords_from, d_coords_to, d_offsets, constant, inverse);
    check_cuda(hipDeviceSynchronize(), "crop");
}

void gpu_crop_scale(Tensor *A, Tensor *B, vector<int> coords_from, vector<int> coords_to, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(B);

    // Copy vector from host to device
    int *d_coords_from; hipMalloc((int**)&d_coords_from, coords_from.size()*sizeof(int));
    hipMemcpy(d_coords_from, coords_from.data(), coords_from.size()*sizeof(int), hipMemcpyHostToDevice);

    // Copy vector from host to device
    int *d_coords_to; hipMalloc((int**)&d_coords_to, coords_to.size()*sizeof(int));
    hipMemcpy(d_coords_to, coords_to.data(), coords_to.size()*sizeof(int), hipMemcpyHostToDevice);

    crop_scale<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_coords_from, d_coords_to, wrapping_mode, constant);
    check_cuda(hipDeviceSynchronize(), "crop_scale");
}


void gpu_pad(Tensor *A, Tensor *B, vector<int> pads){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    int padt = pads[0];
    int padl = pads[3];

    gpu_pad<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], padt, padl);
    check_cuda(hipDeviceSynchronize(), "gpu_pad");
}

void gpu_pad_back(Tensor *A, Tensor *B, vector<int> pads){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    int padt = pads[0];
    int padl = pads[3];

    gpu_pad_back<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], padt, padl);
    check_cuda(hipDeviceSynchronize(), "gpu_pad_back");
}

void gpu_shift_random(Tensor *A, Tensor *B, vector<float> factor_x, vector<float> factor_y, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    float *d_factor_x; hipMalloc((float**)&d_factor_x, 2*sizeof(float));
    hipMemcpy(d_factor_x, factor_x.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Copy vector from host to device
    float *d_factor_y; hipMalloc((float**)&d_factor_y, 2*sizeof(float));
    hipMemcpy(d_factor_y, factor_y.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Generate random numbers
    int N_rnd = A->shape[0] * 2;  // Batch x dims (x, y)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    shift_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], d_factor_x, d_factor_y, wrapping_mode, constant, d_rnd);
    check_cuda(hipDeviceSynchronize(), "shift_random");

     // Free memory
    hipFree(d_factor_x);
    hipFree(d_factor_y);
    hipFree(d_rnd);
}

void gpu_rotate_random(Tensor *A, Tensor *B, vector<float> factor, vector<int> offset_center, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    float *d_factor; hipMalloc((float**)&d_factor, 2*sizeof(float));
    hipMemcpy(d_factor, factor.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Compute angle (radians) and center
    float side_a = A->shape[2]/2.0f;
    float side_b = A->shape[3]/2.0f;
    int center[2] = {(int)side_a+offset_center[0], (int)side_b+offset_center[1]};
    

    // Copy vector from host to device
    int *d_center; hipMalloc((int**)&d_center, 2*sizeof(int));
    hipMemcpy(d_center, center, 2*sizeof(int), hipMemcpyHostToDevice);

    // Generate random numbers
    int N_rnd = A->shape[0] * 1;  // Batch x dims (angle)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    rotate_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], d_factor, d_center, wrapping_mode, constant, d_rnd);
    check_cuda(hipDeviceSynchronize(), "rotate_random");

    // Free memory
    hipFree(d_center);
    hipFree(d_factor);
    hipFree(d_rnd);

}

void gpu_scale_random(Tensor *A, Tensor *B, vector<float> factor, int wrapping_mode, float constant, int transformation_mode){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    float *d_factor; hipMalloc((float**)&d_factor, 2*sizeof(float));
    hipMemcpy(d_factor, factor.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Generate random numbers
    int N_rnd = A->shape[0] * 1;  // Batch x dims (scale)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    scale_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_factor, wrapping_mode, constant, transformation_mode, d_rnd);
    check_cuda(hipDeviceSynchronize(), "scale_random");

    // Free memory
    hipFree(d_factor);
    hipFree(d_rnd);
}

void gpu_flip_random(Tensor *A, Tensor *B, int axis){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Generate random numbers
    int N_rnd = A->shape[0] * 1;  // Batch x dims (apply)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    flip_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], axis, d_rnd);
    check_cuda(hipDeviceSynchronize(), "flip_random");

    // Free memory
    hipFree(d_rnd);
}

void gpu_crop_random(Tensor *A, Tensor *B){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Generate random numbers
    int N_rnd = A->shape[0] * 2;  // Batch x dims (x, y)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    crop_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_rnd);
    check_cuda(hipDeviceSynchronize(), "crop_random");

    // Free memory
    hipFree(d_rnd);
}


void gpu_crop_scale_random(Tensor *A, Tensor *B, vector<float> factor, int wrapping_mode, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    float *d_factor; hipMalloc((float**)&d_factor, 2*sizeof(float));
    hipMemcpy(d_factor, factor.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Generate random numbers
    int N_rnd = A->shape[0] * 3;  // Batch x dims (scale, x, y)
    float* d_rnd = gpu_get_uniforms(N_rnd);
    
    setDims(B);
    crop_scale_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_factor, wrapping_mode, constant, d_rnd);
    check_cuda(hipDeviceSynchronize(), "crop_scale_random");

    // Free memory
    hipFree(d_factor);
    hipFree(d_rnd);
}



void gpu_cutout_random(Tensor *A, Tensor *B, vector<float> factor_x, vector<float> factor_y, float constant){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    float *d_factor_x; hipMalloc((float**)&d_factor_x, 2*sizeof(float));
    hipMemcpy(d_factor_x, factor_x.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Copy vector from host to device
    float *d_factor_y; hipMalloc((float**)&d_factor_y, 2*sizeof(float));
    hipMemcpy(d_factor_y, factor_y.data(), 2*sizeof(float), hipMemcpyHostToDevice);

    // Generate random numbers
    int N_rnd = A->shape[0] * 4;  // Batch x dims (w, h, x, y)
    float* d_rnd = gpu_get_uniforms(N_rnd);

    setDims(B);
    cutout_random<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->shape[2], B->shape[3], d_factor_x, d_factor_y, constant, d_rnd);
    check_cuda(hipDeviceSynchronize(), "cutout_random");

    // Free memory
    hipFree(d_factor_x);
    hipFree(d_factor_y);
    hipFree(d_rnd);
}

void gpu_scale3d(Tensor *A, Tensor *B, vector<int> new_shape, int wrapping_mode, float constant, int transformation_mode){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    int *d_new_shape; hipMalloc((int**)&d_new_shape, new_shape.size()*sizeof(int));
    hipMemcpy(d_new_shape, new_shape.data(), new_shape.size()*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    scale3d<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], A->shape[4], B->shape[2], B->shape[3], B->shape[4], d_new_shape, wrapping_mode, constant, transformation_mode);
    check_cuda(hipDeviceSynchronize(), "scale3d");
}

void gpu_scale3d_back(Tensor *A, Tensor *B, vector<int> new_shape, int wrapping_mode, float constant, int transformation_mode){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy vector from host to device
    int *d_new_shape; hipMalloc((int**)&d_new_shape, new_shape.size()*sizeof(int));
    hipMemcpy(d_new_shape, new_shape.data(), new_shape.size()*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    scale3d_back<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], A->shape[4], B->shape[2], B->shape[3], B->shape[4], d_new_shape, wrapping_mode, constant, transformation_mode);
    check_cuda(hipDeviceSynchronize(), "scale3d_back");
}
