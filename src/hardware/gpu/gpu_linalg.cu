#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <thrust/device_ptr.h>
//#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"



// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct sqr
{
    __host__ __device__
    T operator()(const T& x) const {
        return x * x;
    }
};

float gpu_norm(Tensor *A, string ord){
    int device=A->gpu_device;
    hipSetDevice(device);

    if (ord=="fro"){
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(A->ptr);

        // setup arguments
        sqr<float>        unary_op;
        thrust::plus<float> binary_op;
        float init = 0;

        float abs_sum_sqr = thrust::transform_reduce(dev_ptr, dev_ptr + A->size, unary_op, init, binary_op);
        float norm = std::sqrt(abs_sum_sqr);

        return norm;
    }else{
        msg("Not yet implemented", "gpu_norm");
    }

    return 0.0f;
}


void gpu_norm(Tensor *A, Tensor *B, ReduceDescriptor2 *rd, string ord){
    int device=A->gpu_device;
    hipSetDevice(device);

    gpu_initialize_rd(rd, A, B, true); // Walk through the source tensor
    setDims(B);

    if (ord=="fro") {
        gpu_norm_fro<<<dimGrid, dimBlock>>>(A->ptr, B->ptr, rd->gpu_addresses, B->size, rd->size_reduction);
    }else{
        msg("Not yet implemented", "gpu_norm");
    }

    check_cuda(hipDeviceSynchronize(),"reduce_norm");
}
