/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <cstdio>
#include <string>
#include <stdexcept>
#include <iostream>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"

// CUDA, NVIDIA compute capabilities:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
// -----------------------------------------------------------------
//                      GRID
// Maximum dimensionality of grid of thread blocks:	3
// Maximum x-dimension of a grid of thread blocks	(2^31)-1
// Maximum y- or z-dimension of a grid of thread blocks: 65535
//                   THREAD BLOCK
// Maximum dimensionality of thread block:	3
// Maximum x- or y-dimension of a block:	1024
// Maximum z-dimension of a block:	64
//
// Maximum number of threads per block:	1024
// -----------------------------------------------------------------

hipblasHandle_t hcublas[64];
hiprandGenerator_t random_generator[64];
hipblasStatus_t bstatus;
hiprandStatus_t rstatus;
#ifdef cCUDNN
hipdnnStatus_t dstatus;
hipdnnHandle_t hdnn[64];
#endif

static const char *_curandGetErrorEnum(hiprandStatus_t error){
    switch (error)
    {
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";

        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";

        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";


        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";


        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";


        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";


        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";


        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        default:
            std::string text = "unknown hiprand error: " + std::to_string(error) + " | (_curandGetErrorEnum)";
            throw std::invalid_argument(text);
    }

}

void check_cublas(hipblasStatus_t status, const char *f)
{
    if ( status!=  HIPBLAS_STATUS_SUCCESS)
    {
        std::string text = "error in cublas execution in " + std::string(f) + " | (check_cublas)";
        throw std::runtime_error(text);
    }
}

void check_curand(hiprandStatus_t status, const char *f)
{
    if ( status!=  HIPRAND_STATUS_SUCCESS)
    {
        std::string text = "error in hiprand execution in " + std::string(_curandGetErrorEnum(status)) + " | (check_curand)";
        throw std::runtime_error(text);
    }
}


void check_cuda(hipError_t err,const char *msg)
{
    if(err!=hipSuccess)
    {
        std::string error_type = hipGetErrorString(err);
        std::string text = "[CUDA ERROR]: " + error_type + " ("+ std::to_string(err) + ") raised in " + std::string(msg) + " | (check_cuda)";
        throw std::runtime_error(text);
    }

}

#ifdef cCUDNN

void check_cudnn(hipdnnStatus_t status, const char *msg, const char *file)
{
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::string error_type = hipdnnGetErrorString(status);
        std::string text = "[CUDNN ERROR]: " + error_type + " ("+ std::to_string(status) + ") raised in " + std::string(msg) + " at " + std::string(file) + " file | (check_cudnn)";
        throw std::runtime_error(text);
    }
}

#endif
void gpu_set_device(int device)
{
    hipSetDevice(device);
}


void gpu_init(int device)
{

    int nDevices;
    hipGetDeviceCount(&nDevices);

    if (device>nDevices)
    {
        std::string text = "GPU " + std::to_string(device) + " not available. Number of available GPUs is " + std::to_string(nDevices) + ". Further information running nvidia-smi  | (gpu_init)";
        throw std::runtime_error(text);
    }

    fprintf(stderr,"Selecting GPU device %d\n",device);
    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,device);

    fprintf(stderr,"EDDLL is running on GPU device %d, %s\n",device,prop.name);


    /// CUBLAS
    bstatus=hipblasCreate(&(hcublas[device]));
    // try to init cublas several times
    int i=0;
    while ((bstatus!=  HIPBLAS_STATUS_SUCCESS)&&(i<10)) {
        bstatus=hipblasCreate(&(hcublas[device]));
        i++;
        fprintf(stderr,".\n");
    }

    if ( bstatus!=  HIPBLAS_STATUS_SUCCESS)
    {
        std::string text = "problem in cublas create (gpu_init)";
        throw std::runtime_error(text);
    }
    fprintf(stderr,"CuBlas initialized on GPU device %d, %s\n",device,prop.name);

    bstatus = hipblasSetAtomicsMode(hcublas[device],HIPBLAS_ATOMICS_NOT_ALLOWED);
    if ( bstatus!=  HIPBLAS_STATUS_SUCCESS)
    {
        std::string text = "problem in cublas execution getting: NOT IMPLEMENTED |  (gpu_init)";
        throw std::runtime_error(text);
    }

    // CURAND
    rstatus=hiprandCreateGenerator(&(random_generator[device]),HIPRAND_RNG_PSEUDO_MRG32K3A);
    if (rstatus != HIPRAND_STATUS_SUCCESS)
    {
        std::string text = "error creating random numbers on gpu | (gpu_init)";
        throw std::runtime_error(text);
    }
    rstatus=hiprandSetPseudoRandomGeneratorSeed(random_generator[device],1234);

    if (rstatus != HIPRAND_STATUS_SUCCESS) {
        std::string text = "error setting the seed for program | (gpu_init)";
        throw std::runtime_error(text);
    }
    fprintf(stderr,"CuRand initialized on GPU device %d, %s\n",device,prop.name);
#ifdef cCUDNN
    // CUDNN
    dstatus=hipdnnCreate(&hdnn[device]);
    if (dstatus != HIPDNN_STATUS_SUCCESS) {
        std::string text = "problem in cudnn create (gpu_init)";
        throw std::runtime_error(text);
    }

    fprintf(stderr,"CuDNN initialized on GPU device %d, %s\n",device,prop.name);

#endif

}


float* gpu_create_tensor(int dev,int size)
{
    float* devicePointer;
    hipSetDevice(dev);
    check_cuda(hipMalloc((void**)&devicePointer,size*sizeof(float)),"create_tensor");
    return devicePointer;
}


void gpu_delete_tensor(int dev, float* p)
{
    hipSetDevice(dev);
    check_cuda(hipFree(p),"delete_tensor");
}

void gpu_delete_tensor_int(int dev, int* p)
{
    hipSetDevice(dev);
    check_cuda(hipFree(p),"delete_tensor_int");
}

int gpu_devices()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}
