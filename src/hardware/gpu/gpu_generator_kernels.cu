#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "eddl/hardware/gpu/gpu_kernels.h"


/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init((unsigned int)clock64(), /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void random_uniform(hiprandState_t* states, float* numbers) {
  /* hiprand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand_uniform(&states[blockIdx.x]);
}
