#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "eddl/hardware/gpu/nn/gpu_tensor_nn_kernels.h"
#include "eddl/hardware/gpu/gpu_kernels.h"

__global__ void accuracy(float* T, float* N,float* acc,long int cols, long int total_ops, int* MC_err){

long int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;
long int result_t=T[thread_id_x*cols];
float result_n=N[thread_id_x*cols];

long int row_max_t=0;
long int row_max_n=0;

long int aux_t;
float aux_n;
if (thread_id_x < total_ops)
{
  for(long int i=1;i<cols;i++)
  {
   aux_t=T[thread_id_x*cols+i];
   aux_n=N[thread_id_x*cols+i];

	if (aux_t>result_t)
	 {
  		result_t=aux_t;
      row_max_t=i;
   }
  if (aux_n>result_n)
	 {
		result_n=aux_n;
    row_max_n=i;
   }
  }

  acc[thread_id_x]=row_max_t;
  atomicAdd(MC_err,(long int)(row_max_t==row_max_n));
}

}

__global__ void bin_accuracy(float* T, float* N, int size, int* acc){

long int thread_id_x = threadIdx.x + blockIdx.x*blockDim.x;

if (thread_id_x < size)
{
  if ((T[thread_id_x ]==1.0)&&(N[thread_id_x]>0.5))
    atomicAdd(acc,1);
  else if ((T[thread_id_x ]==0.0)&&(N[thread_id_x]<=0.5))
    atomicAdd(acc,1);

}

}
