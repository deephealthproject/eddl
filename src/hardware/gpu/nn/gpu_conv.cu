#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/nn/gpu_tensor_nn.h"
#include "eddl/hardware/gpu/nn/gpu_tensor_nn_kernels.h"

#include "eddl/hardware/gpu/gpu_hw.h"
#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"

#ifdef cCUDNN
#define cuDNN_GPUS 8
void * shared_workspace[cuDNN_GPUS]; 
size_t workspace_size[cuDNN_GPUS]={0,0,0,0,0,0,0,0};

int allocate_workspace(size_t size, int dev){
    if (size <= workspace_size[dev]){
        return 0;
    }
    else {
        workspace_size[dev] = size;
        hipFree(shared_workspace[dev]);
        return hipMalloc((void **) &shared_workspace[dev], size);
    }
}

//Template created for ConvolDescriptor and ConvolDescriptor3D
template <class condesc>
void cuDNN_environment_initialization(condesc *D){

  int device=D->I->gpu_device;
  hipSetDevice(device);

  int requestedAlgoCount;
  check_cudnn(cudnnGetConvolutionForwardAlgorithmMaxCount( hdnn[device], &requestedAlgoCount),
                                                                "cudnnGetConvolutionForwardAlgorithmMaxCount",__FILE__);

  int returnedAlgoCount;
  hipdnnConvolutionFwdAlgoPerf_t * perfResults = new hipdnnConvolutionFwdAlgoPerf_t [requestedAlgoCount];
  check_cudnn(hipdnnFindConvolutionForwardAlgorithm( hdnn[device], D->xDesc, D->wDesc, D->convolution_descriptor, D->yDesc,
              requestedAlgoCount, &returnedAlgoCount, perfResults),"hipdnnFindConvolutionForwardAlgorithm",__FILE__);

  int aux_alg = 0;
  size_t size;
  do{
      D->fwd_algorithm = perfResults[aux_alg].algo;

      check_cudnn(hipdnnGetConvolutionForwardWorkspaceSize(hdnn[device],D->xDesc, D->wDesc,
                                                              D->convolution_descriptor,  D->yDesc,
                                                              D->fwd_algorithm, &size),
                                                        "hipdnnGetConvolutionForwardWorkspaceSize",__FILE__);
      aux_alg++;
  }
  while(allocate_workspace(size,device));
  //BWD environment
  requestedAlgoCount = 0;

  check_cudnn(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(
              hdnn[device], &requestedAlgoCount),"cudnnGetConvolutionBackwardFilterAlgorithmMaxCount",__FILE__);
  returnedAlgoCount = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t * perfResultsbwf = new hipdnnConvolutionBwdFilterAlgoPerf_t [requestedAlgoCount];

  check_cudnn(hipdnnFindConvolutionBackwardFilterAlgorithm(hdnn[device], D->xDesc, D->yDesc,
                                                        D->convolution_descriptor, D->wDesc, requestedAlgoCount,
                                                        &returnedAlgoCount, perfResultsbwf),
                                                        "hipdnnFindConvolutionBackwardFilterAlgorithm",__FILE__);
  aux_alg = 0;
  size = 0;
  do{
     D->bwd_filter_algorithm = perfResultsbwf[aux_alg].algo;

    check_cudnn(hipdnnGetConvolutionBackwardFilterWorkspaceSize(hdnn[device],D->xDesc, D->yDesc,
                                                              D->convolution_descriptor,  D->wDesc,
                                                              D->bwd_filter_algorithm, &size),
                                                  "hipdnnGetConvolutionBackwardFilterWorkspaceSize",__FILE__);
    aux_alg++;
  }
   while(allocate_workspace(size,device));

  check_cudnn(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(hdnn[device], &requestedAlgoCount),
                    "cudnnGetConvolutionBackwardDataAlgorithmMaxCount", __FILE__);
  returnedAlgoCount=0;
  hipdnnConvolutionBwdDataAlgoPerf_t * perfResults_d = new hipdnnConvolutionBwdDataAlgoPerf_t [requestedAlgoCount];

  check_cudnn(hipdnnFindConvolutionBackwardDataAlgorithm(hdnn[device], D->wDesc, D->yDesc,
                                                        D->convolution_descriptor, D->xDesc, requestedAlgoCount,
                                                        &returnedAlgoCount, perfResults_d),
                                             "(hipdnnFindConvolutionBackwardDataAlgorithm",__FILE__);
  aux_alg = 0;
  size=0;
  do{
      D->bwd_data_algorithm =  perfResults_d[aux_alg].algo;

      check_cudnn(hipdnnGetConvolutionBackwardDataWorkspaceSize(hdnn[device],D->wDesc, D->yDesc,
                                                              D->convolution_descriptor,  D->xDesc,
                                                              D->bwd_data_algorithm, &size),
                                             "hipdnnGetConvolutionBackwardDataWorkspaceSize",__FILE__);
      aux_alg++;
  }
  while(allocate_workspace(size,device));


}

#endif

void gpu_im2col(ConvolDescriptor *D, int col2im){
  int device=D->I->gpu_device;
  hipSetDevice(device);

  setDims(D->gpuI)
  dimGrid.x*=D->I->shape[0];

  if (col2im)
    gpu_im2col_k<<<dimGrid,dimBlock>>>(D->ID->ptr, D->gpuI->ptr,D->I->shape[0],D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,1);
  else
    gpu_im2col_k<<<dimGrid,dimBlock>>>(D->I->ptr, D->gpuI->ptr,D->I->shape[0],D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,0);

  check_cuda(hipDeviceSynchronize(),"gpu_im2col");

}

void gpu_im2col_low(ConvolDescriptor *D, int col2im,int b){
  int device=D->I->gpu_device;
  hipSetDevice(device);

  setDims(D->gpuI)

  if (col2im)
    gpu_im2col_k_low<<<dimGrid,dimBlock>>>(D->ID->ptr, b, D->gpuI->ptr,D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,1);
  else
    gpu_im2col_k_low<<<dimGrid,dimBlock>>>(D->I->ptr, b, D->gpuI->ptr,D->ir,D->ic,D->iz,D->K->ptr,D->nk,D->kr,D->kc,D->O->ptr,D->r,D->c,D->sr,D->sc,D->padrt,D->padrb,D->padcl,D->padcr,0);

  check_cuda(hipDeviceSynchronize(),"gpu_im2col");

}




void gpu_conv2D(ConvolDescriptor *D) {

  int device=D->I->gpu_device;
  hipSetDevice(device);

#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;
  D->gpuK->ptr=D->K->ptr;
  D->gpuO->ptr=D->O->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;


  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuO->ptr+=osize) {
      gpu_im2col_low(D,0,b);
      gpu_mult2D(D->gpuK,0,D->gpuI,1,D->gpuO,0);
    }
  }
  else {

    gpu_im2col(D,0);
    if (D->mem_level==0) {
      gpu_mult2D(D->gpuK,0,D->gpuIB,1,D->gpuOB,0);
      setDims(D->O);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->gpuOB->ptr, D->O->ptr, D->O->shape[0], D->z, D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");
    }
    else {
      gpu_im2col(D,0);
      for(int b=0;b<D->I->shape[0];b++,D->gpuO->ptr+=osize,D->gpuI->ptr+=isize)
        gpu_mult2D(D->gpuK,0,D->gpuI,1,D->gpuO,0);
    }

  }
#else
  // FWD environment
  float alpha = 1.0f;
  float beta = 0.0f;
  if (D->cudnn_env_init < 0){
      D->cudnn_env_init = 1;
      cuDNN_environment_initialization<ConvolDescriptor>(D);
  }
  check_cudnn(hipdnnConvolutionForward( hdnn[device], &alpha, D->xDesc, D->I->ptr,
                                       D->wDesc, D->K->ptr,
                                       D->convolution_descriptor, D->fwd_algorithm,
                                       shared_workspace[device], workspace_size[device],
                                       &beta, D->yDesc, D->O->ptr),"hipdnnConvolutionForward",__FILE__);
#endif
  if (D->use_bias) {
#ifndef cCUDNN
    int size=D->bias->shape[0];
    for(int i=0;i<size;i+=1024) {
      int s=min(1024,size-i);
      gpu_addbias_k<<<D->O->shape[0],s>>>(D->O->ptr, D->O->shape[0], D->r,D->c,D->nk,D->bias->ptr,i);
      check_cuda(hipDeviceSynchronize(),"gpu_addbias");
    }
#else
    check_cudnn(hipdnnAddTensor(hdnn[device], &alpha, D->bDesc, D->bias->ptr,
                               &alpha, D->yDesc, D->O->ptr),"hipdnnAddTensor",__FILE__);
#endif
  }


}


void gpu_conv2D_grad(ConvolDescriptor *D){

  int device=D->I->gpu_device;

  hipSetDevice(device);
  float alpha=1.0;
  float beta = 0.0;
#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;

  D->gpugK->ptr=D->gK->ptr;
  D->gpuD->ptr=D->D->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;

  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize){
      gpu_im2col_low(D,0,b);
      gpu_mult2D(D->gpuD,0,D->gpuI,0,D->gpugK,1);
    }
  }
  else {
    if (D->mem_level==0) {
      setDims(D->D);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->D->ptr, D->gpuOB->ptr, D->z, D->O->shape[0], D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");

      gpu_mult2D(D->gpuOB,0,D->gpuIB,0,D->gpugK,1);
    }
    else {
      for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize,D->gpuI->ptr+=isize)
        gpu_mult2D(D->gpuD,0,D->gpuI,0,D->gpugK,1);
    }
  }
#else
        check_cudnn(hipdnnConvolutionBackwardFilter(hdnn[device], &alpha,
                                      D->xDesc, D->I->ptr,
                                      D->yDesc, D->D->ptr, D->convolution_descriptor,
                                      D->bwd_filter_algorithm,
                                      shared_workspace[device], workspace_size[device],
                                      &beta, D->wDesc, D->gK->ptr),"hipdnnConvolutionBackwardFilter",__FILE__);

#endif
  if (D->use_bias) {
#ifndef cCUDNN
    int size=D->bias->shape[0];
    for(int i=0;i<size;i+=1024) {
      int s=min(1024,size-i);
      gpu_deltabias_k<<<D->D->shape[0],s>>>(D->D->ptr, D->D->shape[0], D->r,D->c,D->nk,D->gbias->ptr,i);
      check_cuda(hipDeviceSynchronize(),"gpu_deltabias");
    }
#else
      check_cudnn(hipdnnConvolutionBackwardBias(hdnn[device], &alpha, D->yDesc, D->D->ptr,
                                               &beta, D->bDesc, D->gbias->ptr),"hipdnnConvolutionBackwardBias",__FILE__);
#endif

  }


}


void gpu_conv2D_back(ConvolDescriptor *D){


  int device=D->I->gpu_device;
  hipSetDevice(device);
#ifndef cCUDNN
  int osize=D->z*D->r*D->c;
  int isize=D->kz*D->kr*D->kc*D->r*D->c;
  D->gpuK->ptr=D->K->ptr;
  D->gpuD->ptr=D->D->ptr;
  D->gpuI->ptr=D->gpuIB->ptr;


  if (D->mem_level>1) {
    for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize) {
        gpu_mult2D(D->gpuD, 1, D->gpuK, 0, D->gpuI, 0);
        gpu_im2col_low(D,1,b);
    }
  }
  else {
    if (D->mem_level==0) {
      setDims(D->D);
      gpu_traspose_batch_depth<<<dimGrid,dimBlock>>>(D->D->ptr, D->gpuOB->ptr,  D->z, D->O->shape[0],D->r, D->c);
      check_cuda(hipDeviceSynchronize(),"gpu_batch_depth");

      gpu_mult2D(D->gpuOB, 1, D->gpuK, 0, D->gpuIB, 0);
      D->gpuI->ptr=D->gpuIB->ptr;
      gpu_im2col(D,1);
    }
    else{
      for(int b=0;b<D->I->shape[0];b++,D->gpuD->ptr+=osize,D->gpuI->ptr+=isize) {
          gpu_mult2D(D->gpuD, 1, D->gpuK, 0, D->gpuI, 0);
      }
      D->gpuI->ptr=D->gpuIB->ptr;
      gpu_im2col(D,1);
    }
  }
#else
    float alpha = 1.0f;
    float beta = 0.0f;
    check_cudnn(hipdnnConvolutionBackwardData(hdnn[device], &alpha, D->wDesc, D->K->ptr,
                                             D->yDesc, D->D->ptr,
                                             D->convolution_descriptor, D->bwd_data_algorithm,
                                             shared_workspace[device], workspace_size[device],
                                             &beta, D->xDesc, D->ID->ptr),"hipdnnConvolutionBackwardData",__FILE__);
#endif

}


void gpu_conv3D(ConvolDescriptor3D *D){
 int device=D->I->gpu_device;
  hipSetDevice(device);
#ifdef cCUDNN
 // FWD environment
  float alpha = 1.0f;
  float beta = 0.0f;
  if (D->cudnn_env_init < 0){
      D->cudnn_env_init = 1;
      cuDNN_environment_initialization<ConvolDescriptor3D>(D);
  }

 check_cudnn(hipdnnConvolutionForward( hdnn[device], &alpha, D->xDesc, D->I->ptr,
                                       D->wDesc, D->K->ptr,
                                       D->convolution_descriptor, D->fwd_algorithm,
                                       shared_workspace[device], workspace_size[device],
                                       &beta, D->yDesc, D->O->ptr),"hipdnnConvolutionForward",__FILE__);
  if (D->use_bias) {
    check_cudnn(hipdnnAddTensor(hdnn[device], &alpha, D->bDesc, D->bias->ptr,
                               &alpha, D->yDesc, D->O->ptr),"hipdnnAddTensor",__FILE__);
  }


#endif

}

void gpu_conv3D_grad(ConvolDescriptor3D *D){
 int device=D->I->gpu_device;
  hipSetDevice(device);
#ifdef cCUDNN
        float alpha = 1.0f;
        float beta = 0.0f;
        check_cudnn(hipdnnConvolutionBackwardFilter(hdnn[device], &alpha,
                                      D->xDesc, D->I->ptr,
                                      D->yDesc, D->D->ptr, D->convolution_descriptor,
                                      D->bwd_filter_algorithm,
                                      shared_workspace[device], workspace_size[device],
                                      &beta, D->wDesc, D->gK->ptr),"hipdnnConvolutionBackwardFilter",__FILE__);
  if (D->use_bias) {
      check_cudnn(hipdnnConvolutionBackwardBias(hdnn[device], &alpha, D->yDesc, D->D->ptr,
                                               &beta, D->bDesc, D->gbias->ptr),"hipdnnConvolutionBackwardBias",__FILE__);
   }
#endif

}

void gpu_conv3D_back(ConvolDescriptor3D *D){
 int device=D->I->gpu_device;
  hipSetDevice(device);
#ifdef cCUDNN
    float alpha = 1.0f;
    float beta = 0.0f;
    check_cudnn(hipdnnConvolutionBackwardData(hdnn[device], &alpha, D->wDesc, D->K->ptr,
                                             D->yDesc, D->D->ptr,
                                             D->convolution_descriptor, D->bwd_data_algorithm,
                                             shared_workspace[device], workspace_size[device],
                                             &beta, D->xDesc, D->ID->ptr),"hipdnnConvolutionBackwardData",__FILE__);
#endif


}
