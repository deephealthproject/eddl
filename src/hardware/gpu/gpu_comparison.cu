#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"


//#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>


// GPU: Structs for Thrust ********************************************

struct is_positive {
    template <typename T>
    bool __device__ operator()(T v) {
        return v > 0;
    }
};


struct all_close {
    const float rtol;
    const float atol;
    const bool equal_nan;

    all_close(float rtol_, float atol_, bool equal_nan_) : rtol(rtol_), atol(atol_), equal_nan(equal_nan_)  { /* empty */ }

    __host__ __device__ bool operator()(float x, float y) const {
        return fabsf(x - y) <= (atol + rtol * fabsf(y));
    }
};


// CPU: Logic functions: Comparisons
void gpu_isfinite(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isfinite<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isfinite");
}

void gpu_isinf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isinf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isinf");
}

void gpu_isnan(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isnan<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isnan");
}

void gpu_isneginf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isneginf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isneginf");
}

void gpu_isposinf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isposinf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isposinf");
}


// CPU: Logic functions: Comparisons
void gpu_logical_and(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_and<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_and");
}

void gpu_logical_or(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_or<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_or");
}

void gpu_logical_not(Tensor *A, Tensor *B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_not<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_not");
}

void gpu_logical_xor(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_xor<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_xor");
}

// GPU: Logic functions: Truth value testing
bool gpu_all(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(A->ptr);
    return thrust::transform_reduce(thrust::device, dev_ptr, dev_ptr+A->size, is_positive{}, true, thrust::logical_and<bool>{} );
}

bool gpu_any(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(A->ptr);
    return thrust::transform_reduce(thrust::device, dev_ptr, dev_ptr+A->size, is_positive{}, false, thrust::logical_or<bool>{} );
}

bool gpu_allclose(Tensor *A, Tensor *B, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);


    thrust::device_ptr<float> A_dev_ptr = thrust::device_pointer_cast(A->ptr);
    thrust::device_ptr<float> B_dev_ptr = thrust::device_pointer_cast(B->ptr);

    thrust::device_vector<float> temp(A->size);
    thrust::transform(A_dev_ptr, A_dev_ptr+A->size, B_dev_ptr, temp.begin(), all_close(rtol, atol, equal_nan));
    return thrust::reduce(thrust::device, temp.begin(), temp.end(), true, thrust::logical_and<bool>{});
    // I think transform_reduce only supports one input vector
    // return thrust::transform_reduce(thrust::device, A_dev_ptr, A_dev_ptr+A->size, B_dev_ptr, all_close(rtol, atol, equal_nan), true, thrust::logical_and<bool>{} );
}

void gpu_isclose(Tensor *A, Tensor *B, Tensor *C, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isclose<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, rtol, atol, equal_nan, A->size);
    check_cuda(hipDeviceSynchronize(), "isclose");
}

void gpu_greater(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "greater");
}

void gpu_greater(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater");
}

void gpu_greater_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "greater_equal");
}

void gpu_greater_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater_equal");
}

void gpu_less(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "less");
}

void gpu_less(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less");
}

void gpu_less_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "less_equal");
}

void gpu_less_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less_equal");
}

void gpu_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "equal");
}

void gpu_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "equal");
}

void gpu_not_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_not_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "not_equal");
}

void gpu_not_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_not_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "not_equal");
}