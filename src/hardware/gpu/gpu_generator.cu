/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.9
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: November 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"



float* gpu_get_uniforms(int N){
    /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<N, 1>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  float* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(float));

  /* invoke the kernel to get some random numbers */
  random_uniform<<<N, 1>>>(states, gpu_nums);

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  // hipFree(gpu_nums);
  return gpu_nums;
}


void gpu_rand_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

  //gpu_mult_(A, v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

}


void gpu_rand_signed_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_signed_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

  A->mult_(2*v);
  A->sub_(v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

}


void gpu_rand_normal(Tensor *A, float m, float s){
  int device=A->gpu_device;
  hipSetDevice(device);

  if (A->size%2) {
    gpu_fill_(A, 0.0);
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size-1,m,s),"gpu_rand_normal");
  }
  else
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size,m,s),"gpu_rand_normal");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_normal");

}


void gpu_rand_binary(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_binary");

  gpu_mask(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_binary");

}
